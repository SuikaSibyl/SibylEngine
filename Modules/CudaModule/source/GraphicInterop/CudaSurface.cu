#include "hip/hip_runtime.h"
#pragma once

#include "CudaModulePCH.h"
#include "CudaSurface.h"
#include "Sibyl/Graphic/Core/Texture/Image.h"

namespace SIByL
{
	CudaSurface::CudaSurface(const std::string& path)
	{
		Image image(path);

		const int width = image.GetWidth();
		const int height = image.GetHeight();
		unsigned char* data = image.GetData();

		// allocate cuda array in device memory
		hipChannelFormatDesc channelDesc =
			hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
		hipArray* cuArray;
		hipMallocArray(&cuArray, &channelDesc, width, height, hipArraySurfaceLoadStore);

		// Set pitch of the source (the width in memory in bytes of the 2D array pointed
		// to by src, including padding), we dont have any padding
		size_t bytesPerElem = sizeof(uchar4);
		// Copy data located at address h_data in host memory to device memory
		checkCudaErrors(hipMemcpy2DToArray(cuArray, 0, 0, data, width * bytesPerElem,
			width * bytesPerElem, height, hipMemcpyHostToDevice));

		// specify texture
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = cuArray;

		// !FAILS! create texture object
		checkCudaErrors(hipCreateSurfaceObject(&mSurfaceObject, &resDesc));
	}

	hipSurfaceObject_t CudaSurface::GetSurfaceObject()
	{
		return mSurfaceObject;
	}

	void CudaSurface::RegisterByOpenGLTexture(uint32_t id, uint32_t width, uint32_t height)
	{
		Width = width;
		Height = height;
		hipGraphicsUnregisterResource(cudaBuffer);
		checkCudaErrors(hipGraphicsGLRegisterImage(&cudaBuffer, id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
	}

	void CudaSurface::StartOpenGLMapping()
	{
		checkCudaErrors(hipGraphicsMapResources(1, &cudaBuffer, 0));

		hipArray_t cuArray = nullptr;
		checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&cuArray, cudaBuffer, 0, 0));

		// specify texture
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = cuArray;

		checkCudaErrors(hipCreateSurfaceObject(&mSurfaceObject, &resDesc));
	}

	void CudaSurface::EndOpenGLMapping()
	{
		checkCudaErrors(hipDestroySurfaceObject(mSurfaceObject));
		checkCudaErrors(hipGraphicsUnmapResources(1, &cudaBuffer, 0));
		checkCudaErrors(hipStreamSynchronize(0));
	}
}