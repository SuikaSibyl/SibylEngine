#include "hip/hip_runtime.h"
#pragma once

#include "CudaModulePCH.h"
#include "CudaTexture.h"
#include "Sibyl/Graphic/Core/Texture/Image.h"

namespace SIByL
{
	CudaTexture::CudaTexture(const std::string& path)
	{
		Image image(path);

		const int width = image.GetWidth();
		const int height = image.GetHeight();
		unsigned char* data = image.GetData();

		// allocate cuda array in device memory
		hipChannelFormatDesc channelDesc =
			hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
		hipArray* cuArray;
		hipMallocArray(&cuArray, &channelDesc, width, height);

		// Set pitch of the source (the width in memory in bytes of the 2D array pointed
		// to by src, including padding), we dont have any padding
		size_t bytesPerElem = sizeof(uchar4);
		// Copy data located at address h_data in host memory to device memory
		checkCudaErrors(hipMemcpy2DToArray(cuArray, 0, 0, data, width * bytesPerElem,
			width * bytesPerElem, height, hipMemcpyHostToDevice));

		// specify texture
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = cuArray;

		// specify texture object parameters
		hipTextureDesc texDesc;
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.addressMode[0] = hipAddressModeWrap;
		texDesc.addressMode[1] = hipAddressModeWrap;
		texDesc.filterMode = hipFilterModePoint;
		texDesc.readMode = hipReadModeElementType;
		texDesc.normalizedCoords = 1;

		// !FAILS! create texture object
		checkCudaErrors(hipCreateTextureObject(&mTextureObject, &resDesc, &texDesc, NULL));
	}

	hipTextureObject_t CudaTexture::GetTextureObject()
	{
		return mTextureObject;
	}

	CudaTexture* CudaTexture::CreateFromOpenGLTexture(uint32_t id, uint32_t width, uint32_t height)
	{
		hipGraphicsResource_t  cudaBuffer;

		checkCudaErrors(hipGraphicsGLRegisterImage(&cudaBuffer, id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));
		checkCudaErrors(hipGraphicsMapResources(1, &cudaBuffer, 0));

		hipArray_t* cuArray = nullptr;
		checkCudaErrors(hipGraphicsSubResourceGetMappedArray(cuArray, cudaBuffer, 0, 0));

		hipChannelFormatDesc cuDesc = hipCreateChannelDesc<uchar4>();
		hipMallocArray(cuArray, &cuDesc, width, height);

		//checkCudaErrors(hipMemcpyToArray(cuArray, 0, 0, pResult, width * height * sizeof(uchar4), hipMemcpyDeviceToDevice));

		//// Set pitch of the source (the width in memory in bytes of the 2D array pointed
		//// to by src, including padding), we dont have any padding
		//size_t bytesPerElem = sizeof(uchar4);
		//// Copy data located at address h_data in host memory to device memory
		//checkCudaErrors(hipMemcpy2DToArray(cuArray, 0, 0, data, width * bytesPerElem,
		//	width * bytesPerElem, height, hipMemcpyHostToDevice));

		hipGraphicsUnmapResources(1, &cudaBuffer, 0);

		return nullptr;
	}

}